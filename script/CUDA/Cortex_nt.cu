#include "hip/hip_runtime.h"
#pragma once
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
// MATLAB related
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "mxShowCriticalErrorMessage.c"
#include "Cortexbasic.cuh"
#include "method.cuh"

#define	N_MX	prhs[0]
#define	E0	    prhs[1]
#define	C_MX 	prhs[2]
#define	CU	    prhs[3]
#define	AREA	prhs[4]
#define	K_MX	prhs[5]
#define	METHOD	prhs[6]
#define	VFLAG	prhs[7]
#define	BLOCKSIZE	prhs[8]

#define	RETVAL1	plhs[0]
#define	RETVAL2	plhs[1]

void mexFunction(int nlhs, mxArray * plhs[], int nrhs, const mxArray * prhs[])
{
    // =========================================================================
    // Flag
    // =========================================================================
    bool vflag = *(bool*)mxGetData(VFLAG);
    int blockSize = *(int*)mxGetData(BLOCKSIZE);
    // =========================================================================
    // initial
    // =========================================================================
    mxInitGPU();
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    float time,timePrepare, timeBasic,timeMethod;
    timeBasic = 0;
    timeMethod = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    const mwSize ndim = 1;
    mwSize dims[ndim];  
    // =========================================================================
    // input N,E0
    // =========================================================================
    hipEventRecord(start, 0);
    int N = *(int*)mxGetData(N_MX); 
    mxGPUArray const *e0 = mxGPUCreateFromMxArray(E0);
    float *d_e0 = (float*)mxGPUGetDataReadOnly(e0);
    const mwSize *dim0 = mxGetDimensions(E0);
    const int L = dim0[1]; // nt
    const int N128 = dim0[0]; 
    if(vflag)mexPrintf("N: %d  N128: %d  L: %d\n",N,N128,L);
    // =========================================================================
    // input C
    // =========================================================================
    mxGPUArray const *c = mxGPUCreateFromMxArray(C_MX);
    int Nc = mxGetM(C_MX);
    int * d_c = (int*)mxGPUGetDataReadOnly(c);
    if(vflag)mexPrintf("Nc: %d\n",Nc);
    // =========================================================================
    // input CU
    // =========================================================================
    mxGPUArray const *cu = mxGPUCreateFromMxArray(CU);
    float * d_cu = (float*)mxGPUGetDataReadOnly(cu);
    // =========================================================================
    // K
    // =========================================================================
    int K = *(int*)mxGetData(K_MX); 
    // =========================================================================
    // Method,area
    // =========================================================================
    int method = *(int*)mxGetData(METHOD); 
    mxGPUArray const *area = mxGPUCreateFromMxArray(AREA);
    float *d_area = (float*)mxGPUGetDataReadOnly(area);
    int Narea = mxGetM(AREA);
    if(vflag)mexPrintf("Narea: %d\n",Narea);
    if(vflag){
        if(method>0){
                mexPrintf("E is in %d order.\n",method);
                mexPrintf("The target is weighted by area. \n");
        }
        else{
            mexPrintf("E is in 1 order.\n");
            mexPrintf("The target is the MAX of E.\nArea is useless.\n");
        }
    }
    if(vflag){ 
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        timePrepare = time;
        mexPrintf("prepare time:  %3.3f ms \n",timePrepare);
        mexEvalString("drawnow") ;
    }
    // =========================================================================
    // predefine return b and internal r
    // =========================================================================
    dims[0] = Nc;
    mxGPUArray * b = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (b==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");
    float *d_b = (float*)mxGPUGetData(b);  
    int rSize = N128/blockSize*K;
    dims[0] = rSize;
    mxGPUArray * r = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (r==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");
    float *d_r = (float*)mxGPUGetData(r);
    if(vflag) mexPrintf("rSize:  %d\n", rSize);
    // =========================================================================
    // process
    // =========================================================================
    int Nloop = Nc/K+1;
    int Nlast = Nc % K;
    int loop100 = ceil((float)Nloop/100);
    if(vflag){
        mexPrintf("Nloop:  %d, K: %d \n", Nloop,K);
        mexEvalString("drawnow") ;
    }
    int Ki = 0;
    for (int i = 0; i<Nloop; i++){
        if(vflag)hipEventRecord(start, 0);
        if(i<Nloop-1) Ki = K;
        else Ki = Nlast;
        if (Ki == 0) break;
        int Cbase = i*K;
        int N1 = N128 / blockSize;
        int gridSize = N128/blockSize*Ki;
        if(i==0){
            if(vflag){
                mexPrintf("blockSize:  %d, gridSize: %d\n", blockSize,gridSize);
                mexEvalString("drawnow") ;
            }
        }
        if(i==loop100){
            if(vflag){
                mexPrintf("The time consumption of %d loops is about %3.3f ms \n", loop100, timeBasic + timeMethod);
                mexEvalString("drawnow") ;
            } 
        }
        unsigned sharedMemSize = blockSize*sizeof(float);
        BasicCortex_nt<<<gridSize, blockSize,sharedMemSize>>>(Ki, N128, d_e0, Nc, Cbase, d_c, d_cu, d_r, method,d_area);
        // if(i==0)RETVAL2 = mxGPUCreateMxArrayOnCPU(r);
        hipDeviceSynchronize();   
        if(vflag){ 
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&time, start, stop);
            timeBasic += time;
        }  
        // =====================================================================
        // method
        // ===================================================================== 
        int M = Ki;
        if(vflag)hipEventRecord(start, 0);
        int blockSize1 = 8;
        int gridSize1 = (M + blockSize1 - 1) / blockSize1;
        if(method==0)
            getMax<<<gridSize1,blockSize1>>>(M, N1, d_r, d_b+i*K);  
        else
            getSum<<<gridSize1,blockSize1>>>(M, N1, d_r, d_b+i*K);
        hipDeviceSynchronize();
        if(vflag){ 
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&time, start, stop);
            timeMethod += time;
        }  
    }    
    if(method>0){
        float areaSum;
        hipblasStatus_t = hipblasSasum(cublasHandle, N, d_area, 1, &areaSum);
        if (hipblasStatus_t!= HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage("hipblasSasum in area failed");
        areaSum = 1/areaSum;
        hipblasStatus_t = hipblasSscal(cublasHandle, Nc, &areaSum, d_b, 1);
        if (hipblasStatus_t!= HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage("hipblasSscal in area scale failed");
    }

    RETVAL1 = mxGPUCreateMxArrayOnCPU(b);
    // =========================================================================
    // destroy
    // =========================================================================
    if(vflag){
        mexPrintf("basic2 time:  %3.3f ms \n",timeBasic);
        mexEvalString("drawnow") ;
        mexPrintf("method time:  %3.3f ms \n",timeMethod);
        mexEvalString("drawnow") ;
        mexPrintf("Other brain part with orientation in GPU ends...\n");
    }
    mxGPUDestroyGPUArray(area);
    mxGPUDestroyGPUArray(r);
    mxGPUDestroyGPUArray(c);
    mxGPUDestroyGPUArray(cu);
    mxGPUDestroyGPUArray(e0);
    mxGPUDestroyGPUArray(b);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipblasDestroy(cublasHandle);
}
  