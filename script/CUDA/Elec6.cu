#include <stdlib.h>
#include <stdio.h>
#include <string.h>
/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
// MATLAB related
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "mxShowCriticalErrorMessage.c"
#include "Elec6basic.cuh"
// #include "method.cuh"

#define	N_MX	prhs[0]
#define	E0	    prhs[1]
#define	AIDX	prhs[2]
#define	BIDX	prhs[3]
#define	C_MX	prhs[4]
#define	BETA_MX	prhs[5]
#define	AREA	prhs[6]
#define	METHOD	prhs[7]
#define	K_MX	prhs[8]
#define	VFLAG	prhs[9]
#define	BLOCKSIZE	prhs[10]


#define	RETVAL1	plhs[0]
#define	RETVAL2	plhs[1]

void mexFunction(int nlhs, mxArray * plhs[], int nrhs, const mxArray * prhs[])
{
    // =========================================================================
    // Flag
    // =========================================================================
    bool vflag = *(bool*)mxGetData(VFLAG);
    int blockSize = *(int*)mxGetData(BLOCKSIZE);
    // =========================================================================
    // initial
    // =========================================================================
    mxInitGPU();
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    hipError_t cudaStatus;
    float time,timePrepare, timeBasic,timeMethod;
    timeBasic = 0;
    timeMethod = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    const mwSize ndim = 1;
    mwSize dims[ndim];      
    if(vflag)hipEventRecord(start, 0);
    // =========================================================================
    // input N,E0
    // =========================================================================
    int N = *(int*)mxGetData(N_MX); 
    mxGPUArray const *e0 = mxGPUCreateFromMxArray(E0);
    float *d_e0 = (float*)mxGPUGetDataReadOnly(e0);
    const mwSize *dim0 = mxGetDimensions(E0);
    const int L = dim0[2];
    const int N128 = dim0[0]; 
    if(vflag)mexPrintf("N: %d  N128: %d  L: %d\n",N,N128,L);
    // =========================================================================
    // Aidx,Bidx
    // =========================================================================
    mxGPUArray const *Aidx = mxGPUCreateFromMxArray(AIDX);
    int *d_Aidx = (int*)mxGPUGetDataReadOnly(Aidx);
    int AidxSize = (int)mxGetNumberOfElements(AIDX);
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(AidxConst), d_Aidx, sizeof(int) * AidxSize);
    if (cudaStatus != hipSuccess) mxShowCriticalErrorMessage("hipMemcpyToSymbol Aidx failed");

    mxGPUArray const *Bidx = mxGPUCreateFromMxArray(BIDX);
    int *d_Bidx = (int*)mxGPUGetDataReadOnly(Bidx);
    int BidxSize = (int)mxGetNumberOfElements(BIDX);
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(BidxConst), d_Bidx, sizeof(int) * BidxSize);
    if (cudaStatus != hipSuccess) mxShowCriticalErrorMessage("hipMemcpyToSymbol Bidx failed");
    // =========================================================================
    // input C
    // =========================================================================
    mxGPUArray const *c = mxGPUCreateFromMxArray(C_MX);
    int Nc = mxGetM(C_MX);
    int * d_c = (int*)mxGPUGetDataReadOnly(c);
    if(vflag)mexPrintf("Nc: %d\t",Nc);
    // =========================================================================
    // K
    // =========================================================================
    int K = *(int*)mxGetData(K_MX);  
    // =========================================================================
    // beta
    // =========================================================================
    mxGPUArray const *beta = mxGPUCreateFromMxArray(BETA_MX);
    float* d_beta = (float*)mxGPUGetDataReadOnly(beta);  
    int betaSize = (int)mxGetNumberOfElements(BETA_MX);
    int Nbeta = (int)mxGetM(BETA_MX);
    if(vflag)mexPrintf("Nbeta: %d\n",Nbeta);
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(betaConst), d_beta, sizeof(float) * betaSize);
    if (cudaStatus != hipSuccess) mxShowCriticalErrorMessage("hipMemcpyToSymbol failed");
    // =========================================================================
    // Method,area
    // =========================================================================
    int method = *(int*)mxGetData(METHOD); 
    mxGPUArray const *area = mxGPUCreateFromMxArray(AREA);
    float *d_area = (float*)mxGPUGetDataReadOnly(area);
    if(vflag){
        if(method>0){
                mexPrintf("E is in %d order.\n",method);
                mexPrintf("The target is weighted by area. \n");
        }
        else{
            mexPrintf("E is in 1 order.\n");
            mexPrintf("The target is the MAX of E.\nArea is useless.\n");
        }
    }
    if(vflag){
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        timePrepare = time;
        mexPrintf("prepare time:  %3.3f ms \n",timePrepare);
        mexEvalString("drawnow") ;
    }
    // =========================================================================
    // predefine return and internal val
    // =========================================================================
    dims[0] = Nbeta*Nc;
    mxGPUArray * b = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (b==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");
    float *d_b = (float*)mxGPUGetData(b);  
    int rSize = N128/blockSize*Nbeta*K;
    // int rSize = N128*Nbeta*K;
    dims[0] = rSize;
    mxGPUArray * r = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (r==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");
    float *d_r = (float*)mxGPUGetData(r);
    // =========================================================================
    // loop
    // =========================================================================
    int Nloop = Nc / K+1;
    if(vflag){
        mexPrintf("LoopNum : %d \n", Nloop);
        mexPrintf("Per loop : %d \n", K);
        mexEvalString("drawnow") ;
    }
    int Nlast = Nc % K;
    int Ki = 0;
    for (int i = 0; i<Nloop; i++){
        // int i = 0;
        if(vflag)hipEventRecord(start, 0);
        if(i<Nloop-1) Ki = K;
        else Ki = Nlast;
        if (Ki == 0) break;
        int N1 = N128 / blockSize;
        int gridSize = N128/blockSize*Ki;
        if(i==0){
            if(vflag){
                hipEventRecord(start, 0);
                mexPrintf("blockSize:  %d, gridSize: %d \n", blockSize,gridSize);
                mexEvalString("drawnow") ;
            }
        }
        int Cbase = i*K;
        unsigned sharedMemSize = blockSize*sizeof(float);
        BasicMore2AB<<<gridSize, blockSize, sharedMemSize>>>(N128, d_e0, Nc, Cbase, Ki, d_c, Nbeta, method, d_r,d_area);
        hipDeviceSynchronize();   
        if(vflag){ 
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&time, start, stop);
            timeBasic += time;
        }
        // method
        int M = Ki*Nbeta;
        if(vflag)hipEventRecord(start, 0); 
        int blockSize1 = 8;
        int gridSize1 = (M + blockSize1 - 1) / blockSize1;
        if(method==0)
            getMax<<<gridSize1,blockSize1>>>(N1, M, Ki, Cbase, Nc, d_r, d_b);  
        else
            getSum<<<gridSize1,blockSize1>>>(N1, M, Ki, Cbase, Nc, d_r, d_b);
        hipDeviceSynchronize();
        if(vflag){ 
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&time, start, stop);
            timeMethod += time;
        }   
    }
    if(method>0){
        float areaSum;
        hipblasStatus_t = hipblasSasum(cublasHandle, N, d_area, 1, &areaSum);
        if (hipblasStatus_t!= HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage("hipblasSasum in area failed");
        areaSum = 1/areaSum;
        hipblasStatus_t = hipblasSscal(cublasHandle, Nc*Nbeta , &areaSum, d_b, 1);
        if (hipblasStatus_t!= HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage("hipblasSscal in method 3 failed");
    }
    RETVAL1 = mxGPUCreateMxArrayOnCPU(b);
    // =========================================================================
    // destroy
    // =========================================================================
    if(vflag){
        mexPrintf("In all : \n");
        mexPrintf("basic2 time:  %3.3f ms \n",timeBasic);
        mexPrintf("method time:  %3.3f ms \n",timeMethod);
        mexEvalString("drawnow") ;
    }
    mxGPUDestroyGPUArray(area);
    mxGPUDestroyGPUArray(r);
    mxGPUDestroyGPUArray(b);
    mxGPUDestroyGPUArray(c);
    mxGPUDestroyGPUArray(e0);
    mxGPUDestroyGPUArray(beta);
    mxGPUDestroyGPUArray(Aidx);
    mxGPUDestroyGPUArray(Bidx);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipblasDestroy(cublasHandle);

}
