#include <stdlib.h>
#include <stdio.h>
#include <string.h>
/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
// MATLAB related
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "mxShowCriticalErrorMessage.c"
#include "Elec8basic.cuh"

#define	N_MX	    prhs[0]
#define	E0	        prhs[1]
#define	AIDX	    prhs[2]
#define	BIDX	    prhs[3]
#define	C_MX	    prhs[4]
#define	KC_MX	    prhs[5]
#define	U_MX	    prhs[6]
#define	KU_MX	    prhs[7]
#define	AREA	    prhs[8]
#define	METHOD	    prhs[9]
#define	VFLAG	    prhs[10]
#define	BLOCKSIZE	prhs[11]


#define	RETVAL1	plhs[0]
#define	RETVAL2	plhs[1]

void mexFunction(int nlhs, mxArray * plhs[], int nrhs, const mxArray * prhs[])
{
    // =========================================================================
    // Flag
    // =========================================================================
    bool vflag = *(bool*)mxGetData(VFLAG);
    int blockSize = *(int*)mxGetData(BLOCKSIZE);
    // =========================================================================
    // initial
    // =========================================================================
    mxInitGPU();
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    hipError_t cudaStatus;
    float time,timePrepare, timeBasic,timeMethod;
    timeBasic = 0;
    timeMethod = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    const mwSize ndim = 1;
    mwSize dims[ndim];      
    if(vflag)hipEventRecord(start, 0);
    // =========================================================================
    // input N,E0
    // =========================================================================
    int N = *(int*)mxGetData(N_MX); 
    mxGPUArray const *e0 = mxGPUCreateFromMxArray(E0);
    float *d_e0 = (float*)mxGPUGetDataReadOnly(e0);
    const mwSize *dim0 = mxGetDimensions(E0);
    const int L = dim0[2];
    const int N128 = dim0[0]; 
    if(vflag)mexPrintf("N: %d  N128: %d  L: %d\n",N,N128,L);
    // =========================================================================
    // Aidx,Bidx
    // =========================================================================
    mxGPUArray const *Aidx = mxGPUCreateFromMxArray(AIDX);
    int *d_Aidx = (int*)mxGPUGetDataReadOnly(Aidx);
    int AidxSize = (int)mxGetNumberOfElements(AIDX);
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(AidxConst), d_Aidx, sizeof(int) * AidxSize);
    if (cudaStatus != hipSuccess) mxShowCriticalErrorMessage("hipMemcpyToSymbol Aidx failed");

    mxGPUArray const *Bidx = mxGPUCreateFromMxArray(BIDX);
    int *d_Bidx = (int*)mxGPUGetDataReadOnly(Bidx);
    int BidxSize = (int)mxGetNumberOfElements(BIDX);
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(BidxConst), d_Bidx, sizeof(int) * BidxSize);
    if (cudaStatus != hipSuccess) mxShowCriticalErrorMessage("hipMemcpyToSymbol Bidx failed");
    // =========================================================================
    // C, Kc
    // =========================================================================
    mxGPUArray const *c = mxGPUCreateFromMxArray(C_MX);
    int Nc = mxGetM(C_MX);
    int * d_c = (int*)mxGPUGetDataReadOnly(c);
    int Kc = *(int*)mxGetData(KC_MX);  
    if(vflag)mexPrintf("Nc: %d  Kc: %d\n",Nc,Kc);
    // =========================================================================
    // u, Ku
    // =========================================================================
    mxGPUArray const *u = mxGPUCreateFromMxArray(U_MX);
    int Mu = (int)mxGetM(U_MX);
    int Nu = (int)mxGetN(U_MX);
    float* d_u = (float*)mxGPUGetDataReadOnly(u);  
    int Ku = *(int*)mxGetData(KU_MX);  
    if(vflag)mexPrintf("Nu: %d, Mu: %d, Ku: %d\n",Nu,Mu,Ku);
    // =========================================================================
    // Method,area
    // =========================================================================
    int method = *(int*)mxGetData(METHOD); 
    mxGPUArray const *area = mxGPUCreateFromMxArray(AREA);
    float *d_area = (float*)mxGPUGetDataReadOnly(area);
    if(vflag){
        if(method>0){
                mexPrintf("E is in %d order.\n",method);
                mexPrintf("The target is weighted by area. \n");
        }
        else{
            mexPrintf("E is in 1 order.\n");
            mexPrintf("The target is the MAX of E.\nArea is useless.\n");
        }
    }
    if(vflag){
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        timePrepare = time;
        mexPrintf("prepare time:  %3.3f ms \n",timePrepare);
        mexEvalString("drawnow") ;
    }
    // =========================================================================
    // predefine return b and internal r
    // =========================================================================
    // b
    dims[0] = Nu*Nc;
    mxGPUArray * b = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (b==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");
    float *d_b = (float*)mxGPUGetData(b);  
    // r
    int rSize = N128/blockSize*Ku*Kc;
    dims[0] = rSize;
    mxGPUArray * r = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (r==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");
    float *d_r = (float*)mxGPUGetData(r);

    // =========================================================================
    // loop
    // =========================================================================
    int Loopc = Nc/Kc+1;
    int Loopu = Nu/Ku+1;
    if(vflag){
        mexPrintf("LoopCNum : %d, LoopUNum: %d \n", Loopc,Loopu);
        mexEvalString("drawnow") ;
    }
    int Lastc = Nc % Kc;
    int Lastu = Nu % Ku;
    int Kci,Kui;
    // =========================================================================
    // iuNum
    // =========================================================================
    int UConstMax = 8192/Mu;
    if(UConstMax<Ku) mxShowCriticalErrorMessage("Not enough constant space for Ku!");
    for (int iu = 0; iu<Loopu; iu++){
        if(iu<Loopu-1) Kui = Ku;
        else Kui = Lastu;
        if (Kui == 0) continue;
        int uConstSize = sizeof(float) *Kui*Mu;
        cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(UConst), d_u+iu*Ku*Mu, uConstSize);
        if (cudaStatus != hipSuccess) mxShowCriticalErrorMessage("uConst set failed");
        for (int ic = 0; ic<Loopc; ic++){
            // int ic = 0;
            // int iu = 0;
            if(vflag)hipEventRecord(start, 0);
            // Kci, Kui
            if(ic<Loopc-1) Kci = Kc;
            else Kci = Lastc;
            if (Kci == 0) continue;
            int N1 = N128 / blockSize;
            int gridSize = N128/blockSize*Kci;
             if(ic==0 & iu==0){
                if(vflag){
                    hipEventRecord(start, 0);
                    mexPrintf("blockSize:  %d, gridSize: %d \n", blockSize,gridSize);
                    mexPrintf("Kci:  %d, Kui: %d \n", Kci,Kui);
                    mexEvalString("drawnow") ;
                }
            }
            int basec = ic*Kc;
            int baseu = iu*Ku;
            unsigned sharedMemSize = blockSize*sizeof(float);
            BasicElec8<<<gridSize, blockSize, sharedMemSize>>>(N128, d_e0, Nc, basec, Kci, d_c, Mu, Kui, method, d_r, d_area);
            hipDeviceSynchronize(); 
            // if(iu==0)
            //     RETVAL2 = mxGPUCreateMxArrayOnCPU(r);  
            if(vflag){ 
                hipEventRecord(stop, 0);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&time, start, stop);
                timeBasic += time;
            }
            // method
            int M = Kci*Kui;
            if(vflag)hipEventRecord(start, 0); 
            int blockSize1 = 8;
            int gridSize1 = (M + blockSize1 - 1) / blockSize1;
            
            if(method==0)
                getMax<<<gridSize1,blockSize1>>>(N1, M, Kci, basec, Nc, baseu, d_r, d_b);  
            else
                getSum<<<gridSize1,blockSize1>>>(N1, M, Kci, basec, Nc, baseu, d_r, d_b);
            
                hipDeviceSynchronize();
            if(vflag){ 
                hipEventRecord(stop, 0);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&time, start, stop);
                timeMethod += time;
            }   
        }
    }
    if(method>0){
        float areaSum;
        hipblasStatus_t = hipblasSasum(cublasHandle, N, d_area, 1, &areaSum);
        if (hipblasStatus_t!= HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage("hipblasSasum in area failed");
        areaSum = 1/areaSum;
        hipblasStatus_t = hipblasSscal(cublasHandle, Nc*Nu , &areaSum, d_b, 1);
        if (hipblasStatus_t!= HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage("hipblasSscal in method 3 failed");
    }
    RETVAL1 = mxGPUCreateMxArrayOnCPU(b);
    // =========================================================================
    // destroy
    // =========================================================================
    if(vflag){
        mexPrintf("In all : \n");
        mexPrintf("basic2 time:  %3.3f ms \n",timeBasic);
        mexPrintf("method time:  %3.3f ms \n",timeMethod);
        mexEvalString("drawnow") ;
    }
    mxGPUDestroyGPUArray(area);
    mxGPUDestroyGPUArray(r);
    mxGPUDestroyGPUArray(b);
    mxGPUDestroyGPUArray(c);
    mxGPUDestroyGPUArray(e0);
    mxGPUDestroyGPUArray(u);
    mxGPUDestroyGPUArray(Aidx);
    mxGPUDestroyGPUArray(Bidx);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipblasDestroy(cublasHandle);

}
