#include <stdlib.h>
#include <stdio.h>
#include <string.h>
/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
// MATLAB related
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "mxShowCriticalErrorMessage.c"
#include "ROIbasic.cuh"


#define	N_MX	prhs[0]
#define	E0	    prhs[1]
#define	C_MX	prhs[2]
#define	BETA_MX	prhs[3]
#define	METHOD	prhs[4]
#define	K_MX	prhs[5]
#define	VFLAG	prhs[6]
#define	AREA	prhs[7]
#define	BLOCKSIZE	prhs[8]

#define	RETVAL1	plhs[0]
#define	RETVAL2	plhs[1]

void mexFunction(int nlhs, mxArray * plhs[], int nrhs, const mxArray * prhs[])
{
    bool vflag = *(bool*)mxGetData(VFLAG);
    int method = *(int*)mxGetData(METHOD);
    int blockSize = *(int*)mxGetData(BLOCKSIZE);
    // =========================================================================
    // initial
    // =========================================================================
    mxInitGPU();
    float time,timeBasic,timeMethod;
    timeBasic = 0;
    timeMethod = 0;
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    hipError_t cudaStatus;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    const mwSize ndim = 1;
    mwSize dims[ndim];
    // =========================================================================
    // input N,E0
    // =========================================================================
    int N = *(int*)mxGetData(N_MX); 
    if (vflag)hipEventRecord(start, 0);
    mxGPUArray const *e0 = mxGPUCreateFromMxArray(E0);
    float *d_e0 = (float*)mxGPUGetDataReadOnly(e0);
    const mwSize *dim0 = mxGetDimensions(E0);
    const int L = dim0[2];
    const int N128 = dim0[0]; 
    if (vflag){
        mexPrintf("N: %d  N128: %d  L: %d\n",N,N128,L);
    }
    // =========================================================================
    // input C_MX
    // =========================================================================
    mxGPUArray const *c = mxGPUCreateFromMxArray(C_MX);
    int Nc = mxGetM(C_MX);
    int * d_c = (int*)mxGPUGetDataReadOnly(c);
    if (vflag){
        mexPrintf("Nc: %d\n",Nc);
    }
    // =========================================================================
    // input beta
    // =========================================================================
    mxGPUArray const *beta = mxGPUCreateFromMxArray(BETA_MX);
    float* d_beta = (float*)mxGPUGetDataReadOnly(beta);  
    int Nbeta = (int)mxGetNumberOfElements(BETA_MX);
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(betaConst), d_beta, sizeof(float) * Nbeta);
    if (cudaStatus != hipSuccess) mxShowCriticalErrorMessage("hipMemcpyToSymbol failed");
    // =========================================================================
    // input area,K
    // =========================================================================
    int K = *(int*)mxGetData(K_MX); 
    mxGPUArray const *area = mxGPUCreateFromMxArray(AREA);
    float *d_area = (float*)mxGPUGetDataReadOnly(area);
    // =========================================================================
    // predefine return b and internal r
    // =========================================================================
    dims[0] = Nbeta * Nc;
    mxGPUArray * b = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (b==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");
    float *d_b = (float*)mxGPUGetData(b);
    int rSize = N128/blockSize*K;
    dims[0] = rSize;
    mxGPUArray * r = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (r==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");
    float *d_r = (float*)mxGPUGetData(r);
    // =========================================================================
    // kernel
    // =========================================================================
    if(vflag){
        if(method>0){
                mexPrintf("E is in %d order.\n",method);
                mexPrintf("The target is weighted by area. \n");
        }
        else{
            mexPrintf("E is in 1 order.\n");
            mexPrintf("The target is the MAX of E.\nArea is useless.\n");
        }
    }
    if (vflag){
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        mexPrintf("Prepare time:  %3.3f ms \n", time);
        mexEvalString("drawnow") ;
    }
    int Nloop = Nc/K+1;
    int Nlast = Nc % K;
    int Ki = 0;
    for (int i = 0; i<Nloop; i++){
        if(vflag)hipEventRecord(start, 0);
        if(i<Nloop-1) Ki = K;
        else Ki = Nlast;
        if (Ki == 0) break;
        int Cbase = i*K;
        int N1 = N128 / blockSize;
        int gridSize = N128/blockSize*Ki;
        if(i==0){
            if(vflag){
                mexPrintf("blockSize:  %d, gridSize: %d \n", blockSize,gridSize);
                mexPrintf("Nloop: %d, Ki:  %d\n", Nloop,Ki);
                mexEvalString("drawnow") ;
            }
        }
        unsigned sharedMemSize = blockSize * sizeof(float);
        BasicROI<<<gridSize, blockSize,sharedMemSize>>>(N128, Ki, Nc, d_e0, d_c, Cbase, Nbeta, d_r, method, d_area);
        hipDeviceSynchronize();   
        if(vflag){ 
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&time, start, stop);
            timeBasic += time;
        }
        if(i==0)RETVAL2 = mxGPUCreateMxArrayOnCPU(r); 
        // =====================================================================
        // method
        // =====================================================================
        int M = Ki*Nbeta;
        if(vflag)hipEventRecord(start, 0); 
        int blockSize1 = 8;
        int gridSize1 = (M + blockSize1 - 1) / blockSize1;
        if(method==0)
            getMax<<<gridSize1,blockSize1>>>(N1, M, Ki, Cbase, Nc, d_r, d_b);  
        else
            getSum<<<gridSize1,blockSize1>>>(N1, M, Ki, Cbase, Nc, d_r, d_b);
        hipDeviceSynchronize();
        if (vflag){
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&time, start, stop);
            timeMethod += time;
        }
    }
    if(method>0){
        float areaSum;
        hipblasStatus_t = hipblasSasum(cublasHandle, N, d_area, 1, &areaSum);
        if (hipblasStatus_t!= HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage("hipblasSasum in area failed");
        areaSum = 1/areaSum;
        hipblasStatus_t = hipblasSscal(cublasHandle, Nc*Nbeta , &areaSum, d_b, 1);
        if (hipblasStatus_t!= HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage("hipblasSscal in method 3 failed");
    }
    // =========================================================================
    // output
    // =========================================================================
    RETVAL1 = mxGPUCreateMxArrayOnCPU(b);
    // =========================================================================
    // destroy
    // =========================================================================
    mxGPUDestroyGPUArray(r);
    mxGPUDestroyGPUArray(b);
    mxGPUDestroyGPUArray(e0);
    mxGPUDestroyGPUArray(area);
    mxGPUDestroyGPUArray(c);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipblasDestroy(cublasHandle);
    if(vflag){
        mexPrintf("basic2 time:  %3.3f ms \n",timeBasic);
        mexPrintf("method time:  %3.3f ms \n",timeMethod);
        mexPrintf("ROI gpu part end...\n");
    }
}
  